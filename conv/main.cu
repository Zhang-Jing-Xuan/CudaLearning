
#include <hip/hip_runtime.h>
#include <stdio.h>

static void HandleError(hipError_t err,const char * file,int line){
    if(err!=hipSuccess){
        printf("%s in %s at line %d\n",hipGetErrorString(err),file,line);
        exit(EXIT_FAILURE);
    }
}

#define HANDLE_ERROR(err) (HandleError(err,__FILE__,__LINE__))

int getThreadNum(){
    hipDeviceProp_t prop;
    int count;
    HANDLE_ERROR(hipGetDeviceCount(&count));
    printf("gpu num %d\n",count);
    HANDLE_ERROR(hipGetDeviceProperties(&prop,0));
    printf("max thread num:%d\n",prop.maxThreadsPerBlock);
    printf("max grid dimensions:%d %d %d\n",prop.maxGridSize[0],prop.maxGridSize[1],prop.maxGridSize[2]);
    return prop.maxThreadsPerBlock;
}

__global__ void conv(float *img,float *kernel,float *result,int width,int height,int kernelSize){
    int ti=threadIdx.x;
    int bi=blockIdx.x;
    int id=bi*blockDim.x+ti;
    if(id>=width*height){
        return ;
    }
    int row=id/width,col=id%width;
    for(int i=0;i<kernelSize;i++){
        for(int j=0;j<kernelSize;j++){
            float imgValue=0;
            int curRow=row-kernelSize/2+i;
            int curCol=col-kernelSize/2+j;
            if(curRow<0||curCol<0||curRow>=height||curCol>=width){
                    ;
            }else{
                imgValue=img[curRow*width+curCol];
            }
            result[id]+=kernel[i*kernelSize+j]*imgValue;
        }
    }
}
int main(){
    int width=10;
    int height=10;
    float *img=new float[width*height];
    for(int i=0;i<height;i++){
        for(int j=0;j<width;j++){
            img[j+i*width]=(i+j)%256;
        }
    }

    int kernelSize=3;
    float *kernel=new float[kernelSize*kernelSize];
    for(int i=0;i<kernelSize*kernelSize;i++){
        kernel[i]=i%kernelSize-1;
    }

    float *imgGpu,*kernelGpu,*resultGpu;

    HANDLE_ERROR(hipMalloc((void**)&imgGpu,width*height*sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&kernelGpu,kernelSize*kernelSize*sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&resultGpu,width*height*sizeof(float)));

    HANDLE_ERROR(hipMemcpy(imgGpu,img,width*height*sizeof(float),hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(kernelGpu,kernel,kernelSize*kernelSize*sizeof(float),hipMemcpyHostToDevice));

    int threadNum=getThreadNum();
    int blockNum=(width*height-0.5)/threadNum+1;
    conv<<<blockNum,threadNum>>>(imgGpu,kernelGpu,resultGpu,width,height,kernelSize);

    float *result=new float[width*height];
    HANDLE_ERROR(hipMemcpy(result,resultGpu,width*height*sizeof(float),hipMemcpyDeviceToHost));
    //Visualization
    printf("img:\n");
    for(int i=0;i<10;i++){
        for(int j=0;j<10;j++){
            printf("%2.0f ",img[j+i*width]);
        }
        puts("");
    }
    printf("kernel:\n");
    for(int i=0;i<kernelSize;i++){
        for(int j=0;j<kernelSize;j++){
            printf("%2.0lf ",kernel[i*kernelSize+j]);
        }
        puts("");
    }
    printf("result:\n");
    for(int i=0;i<10;i++){
        for(int j=0;j<10;j++){
            printf("%2.0f ",result[j+i*width]);
        }
        puts("");
    }
    return 0;
}