
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

__global__ void add(int* a, int* b, int* c, int num)
{
	int i = threadIdx.x; //the id of thread
	if(i < num)
	{
		c[i] = a[i] + b[i];
	}
}

int main(void)
{
	// init data
	int num = 10;
	int a[num], b[num], c[num];
	int *a_gpu, *b_gpu, *c_gpu;

	for(int i = 0; i < num; i++)
	{
		a[i] = i;
		b[i] = i * i;
	}

	hipMalloc((void **)&a_gpu, num * sizeof(int));
	hipMalloc((void **)&b_gpu, num * sizeof(int));
	hipMalloc((void **)&c_gpu, num * sizeof(int));

	// copy data
	hipMemcpy(a_gpu, a, num * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(b_gpu, b, num * sizeof(int), hipMemcpyHostToDevice);

	//do 
	add <<<1,num>>>(a_gpu,b_gpu,c_gpu,num);// 1 block,10 thread
	
	// get data
	hipMemcpy(c, c_gpu, num * sizeof(int), hipMemcpyDeviceToHost);

	// visualization
	for(int i = 0; i < num; i++)
	{
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	return 0;
}